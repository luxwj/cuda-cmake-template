
#include <hip/hip_runtime.h>
# include <stdio.h>

__global__ void test_kernel(double *arr) {
    *arr = 3.1415926;
}

int main() {
    double *arr_h, *arr_d;
    size_t arr_size = sizeof(double) * 50;
    arr_h = (double *)malloc(arr_size);
    hipMalloc((void **)&arr_d, arr_size);
    dim3 gridDim = dim3(1, 1, 1);
    dim3 blockDim = dim3(1, 1, 1);

    test_kernel<<<gridDim, blockDim>>>(arr_d);
    hipMemcpy(arr_h, arr_d, arr_size, hipMemcpyDeviceToHost);
    printf("Hello world!\n");
    if (*arr_h == 3.1415926) {
        printf("Hello world!\n");
    }
    return 0;
}